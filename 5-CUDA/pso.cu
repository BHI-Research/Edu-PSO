#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <float.h>
#include <math.h>
#include <string.h>

#include "pso.h"





// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


/**
 * CUDA Kernel Device code
 **/
__global__ void
calc(double *vel,double *pos,double *pos_b/*,double *pos_nb*/,
     double c1, double c2,double w,int step, int numElements,
     double x_hi, double x_lo)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;


    int indice = j * numElements + i;

    if (i < numElements && j< numElements)
    {
        // calculate stochastic coefficients
        double rho1 = c1  *(0.5+/*(float)(i/1e5)+*/(double)(step/1e10));//random_float(0,1); //SE ANULA PORQUE NO PERMITE PARALIZACION
        double rho2 = c2  *(0.5+/*(float)(i/1e5)+*/(double)(step/1e10));//random_float(0,1); //SE ANULA PORQUE NO PERMITE PARALIZACION
        vel[indice] =w*vel[indice]+rho1*(pos_b[indice]-pos[indice])+rho2*(/*pos_nb[indice]*/-pos[indice]);
        pos[indice] = pos[indice] + vel[indice];
        
        if (pos[indice] < x_lo)
        {
            pos[indice] = x_lo;
            vel[indice] = 0;
        } 
        else if (pos[indice] > x_hi)
        {
            pos[indice] = x_hi;
            vel[indice] = 0;
        }
    }
}

/// Functions Propotypes
//Free GPU Memory
bool free_memGPU (double *arr1, double *arr2, double *arr3);
//Check CUDA Errors
bool check (hipError_t error );




// calulate swarm size based on dimensionality
int pso_calc_swarm_size(int dim) {                          //NO SE UTILIZA PORQUE SE INGRESA POR CONSOLA Nro DE PARTICULAS
    int size = 10. + 2. * sqrt(dim);
    return (size > PSO_MAX_SIZE ? PSO_MAX_SIZE : size);
}

// return default pso settings
void pso_set_default_settings(pso_settings_t *settings) {

    // set some default values
    //settings->dim = 2026;         //PARAMETRO INGRESADO EN EL SCRIP (MODIFICA NUMERO DE DIMENCIONES)
    //settings->x_lo = -20;         //PARAMETRO INGRESADO EN EL SCRIP (MODIFICA EL LIMITE INFERIOR DEL RANGO)
    //settings->x_hi = 20;          //PARAMETRO INGRESADO EN EL SCRIP (MODIFICA EL LIMITE SUPERIOR DEL RANGO)
    //settings->goal = 1e-5;        //PARAMETRO INGRESADO EN EL SCRIP (MODIFICA EL UMBRAL DE ERROR)

    //settings->size = pso_calc_swarm_size(settings->dim);  //PARAMETRO INGRESADO EN EL SCRIP (MODIFICA NUMERO DE PARTICULAS)
    settings->print_every = 1000;
    settings->steps = 100000;
    settings->c1 = 1.496;
    settings->c2 = 1.496;
    settings->w_max = PSO_INERTIA;
    settings->w_min = 0.3;

    settings->clamp_pos = 1;
}

float random_float(float min, float max)
{
    float random = ((float) rand()) / (float) RAND_MAX;
    float dif = max - min;
    float range = random * dif;
    return min + range;
}

void pso_solve(pso_obj_fun_t obj_fun, void *obj_fun_params, pso_result_t *solution, pso_settings_t *settings, FILE *file)
{
    //printf("Total particles number: %d\n", settings->size);   //SE ANULA PORQUE AHORA SE IMPRIME AL FINAL

    // Particles
    double pos[settings->size][settings->dim]; // matriz posicion
    double vel[settings->size][settings->dim]; // matriz velocidad
    double pos_b[settings->size][settings->dim]; // matriz mejor  posicion
    double fit[settings->size]; // vector fitness de la particula
    double fit_b[settings->size]; // vector mejor fitness de la particula

    // Swarm
    double pos_nb[settings->size][settings->dim]; // what is the best informed

    // position for each particle
    int comm[settings->size][settings->size]; // communications:who informs who
    int improved; // whether solution->error was improved duringthe last iteration

    int part_id, dim_id, step;
    double a, b; // for matrix initialization
    double rho1, rho2; // random numbers (coefficients)
    double w; // current omega
    void (*inform_fun)(); // neighborhood update function
    double (*calc_inertia_fun)(); // inertia weight update function


    // INITIALIZE SOLUTION
    solution->error = DBL_MAX;

    // SWARM INITIALIZATION
    // for each particle
    for (part_id=0; part_id<settings->size; part_id++)
    {
        // for each dimension
        for (dim_id=0; dim_id<settings->dim; dim_id++)
        {
            // generate two numbers within the specified range
            a = settings->x_lo + (settings->x_hi - settings->x_lo) * (0.5+(double)(part_id/1e5)+(double)(dim_id/1e8));//random_float(0,1); //SE ANULA PORQUE NO PERMITE PARALIZACION
            b = settings->x_lo + (settings->x_hi - settings->x_lo) * (0.5+(double)(part_id/1e5)+(double)(dim_id/1e8));//random_float(0,1); //SE ANULA PORQUE NO PERMITE PARALIZACION
            // initialize position
            pos[part_id][dim_id] = a;
            // best position is the same
            pos_b[part_id][dim_id] = a;
            // initialize velocity
            vel[part_id][dim_id] = (a-b) / 2.;
        }
        // update particle fitness
        fit[part_id] = obj_fun(pos[part_id], settings->dim, obj_fun_params);
        fit_b[part_id] = fit[part_id]; // this is also the personal best
        // update gbest??
        if (fit[part_id] < solution->error)
        {
            // update best fitness
            solution->error = fit[part_id];
            // copy particle pos to gbest vector
            memmove((void *)solution->gbest, (void *)&pos[part_id],sizeof(double) * settings->dim);
        }
    }

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    int numElements = settings->size;
    size_t size = settings->size * settings->dim * sizeof(double);

    //Vectors on GPU Memory
    double *d_vel = NULL;
    double *d_pos = NULL;
    double *d_pos_b = NULL;
    //double *d_pos_nb = NULL;

    hipMalloc((void **)&d_vel, size);
    hipMalloc((void **)&d_pos, size);
    hipMalloc((void **)&d_pos_b, size);
    //hipMalloc((void **)&d_pos_nb, size);

    hipMemcpy(d_vel, vel, size, hipMemcpyHostToDevice);
    hipMemcpy(d_pos, pos, size, hipMemcpyHostToDevice);
    hipMemcpy(d_pos_b, pos_b, size, hipMemcpyHostToDevice);
    //hipMemcpy(d_pos_nb, pos_nb, size, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    dim3 threadsPerBlock(16,16);
    int block=(int) ceil((float)numElements / 16.0);
    dim3 blocksPerGrid(block,block);

    // initialize omega using standard value
    w = PSO_INERTIA;
    // RUN ALGORITHM
    for (step=0; step<settings->steps; step++)
    {

        //hipMemcpy(d_pos, pos, size, hipMemcpyHostToDevice);
        hipMemcpy(d_pos_b, pos_b, size, hipMemcpyHostToDevice);
        // update current step
        settings->step = step;

        // check optimization goal
        if (solution->error <= settings->goal)
        {
            // SOLVED!!
            if (settings->print_every)
                printf("Goal achieved @ step %d (error=%.3e) :-)\n", step, solution->error);
            printf("Best known position: [");
            for (part_id=0; part_id<settings->dim; part_id++) {
                printf("%6.2lf", solution->gbest[part_id]);
            }
            printf("]\n");
            break;
        }

        // the value of improved was just used; reset it
        improved = 0;



        // Operation with the GPU
        calc<<<blocksPerGrid, threadsPerBlock>>>(d_vel, d_pos, d_pos_b/*,d_pos_nb*/,settings->c1,settings->c2,w,step, numElements, settings->x_hi, settings->x_lo);

        //hipMemcpy(vel, d_vel, size, hipMemcpyDeviceToHost);
        hipMemcpy(pos, d_pos, size, hipMemcpyDeviceToHost);
        hipMemcpy(pos_b, d_pos_b, size, hipMemcpyDeviceToHost);
        //hipMemcpy(pos_nb, d_pos_nb, size, hipMemcpyDeviceToHost);





        // update all particles
        for (part_id=0; part_id<settings->size; part_id++)
        {
            // update particle fitness
            fit[part_id] = obj_fun(pos[part_id], settings->dim, obj_fun_params);
            // update personal best position?
            if (fit[part_id] < fit_b[part_id])
            {
                fit_b[part_id] = fit[part_id];
                // copy contents of pos[i] to pos_b[i]
                memmove((void *)&pos_b[part_id], (void *)&pos[part_id],sizeof(double) * settings->dim);
            }
            // update gbest??
            if (fit[part_id] < solution->error)
            {
                improved = 1;
                // update best fitness
                solution->error = fit[part_id];
                // copy particle pos to gbest vector
                memmove((void *)solution->gbest, (void *)&pos[part_id],sizeof(double) * settings->dim);
            }
        }


        if (settings->print_every && (step % settings->print_every == 0))
            printf("Step %d (w=%.2f) :: min err=%.5e\n", step, w, solution->error);

        if(step % (settings->print_every/10) == 0) //IMPRIME POSICIONES DE PARTICULAS EN UN ARCHIVO CADA 100 STEPS
        {
            for (int part_id2=0; part_id2<settings->size; part_id2++)
            {
                for (int dim_id2=0; dim_id2<settings->dim; dim_id2++)
                {
                    fprintf( file, "%f ", pos[part_id2][dim_id2]);
                }
                fprintf( file, "\n");
            }
        }

    }
            free_memGPU(d_vel,d_pos,d_pos_b/*,d_pos_nb*/);

}



bool free_memGPU (double *arr1,double *arr2,double *arr3/*,double *arr4*/){

    // Free device global memory
    hipError_t err;
    err = hipFree(arr1);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        return 0;
    }

    err = hipFree(arr2);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        return 0;
    }

    err = hipFree(arr3);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        return 0;
    }

    /*err = hipFree(arr4);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector D (error code %s)!\n", hipGetErrorString(err));
        return 0;
    }*/
    //printf("Resources free from CUDA Device\n");
    return 1;

}

bool check (hipError_t error ){
    if (error != hipSuccess) return 0;
    //printf ("Error checkeado\n");
    return 1;
}
